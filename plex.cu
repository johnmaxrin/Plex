#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctype.h>
#include <sstream>
#include "helpers/enum.h"
#include "helpers/hostHelp.h"
#include "helpers/global.h"
#include "helpers/helper.h"
#include "fileops/plexFile.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include<time.h>
 


__global__ void init(char *d_str, int *tokens, char *values, int size)
{

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if(idx >= size)
    return;

  char *data = &d_str[idx*MTS];
  char *parseData;
  parseData = parse(data, tokens, values,idx);
  int start = idx*MTS;

  int i=start,j=0;
  while(parseData[j]!='\0')
    values[i++] = parseData[j++];
  

  }


int main(int argc, char *argv[])
{
  int *htokens, *tokens;
  char *hvalues, *values, *d_str;

  // Reading file  
  thrust::host_vector<std::string> h_vec = readFile(argv[1]);

  hipError_t error =  hipMalloc(&d_str, h_vec.size() * sizeof(char) * MTS);
  for (int i = 0; i < h_vec.size(); ++i)
  {
    std::string str = h_vec[i];
    const char *c_str = str.c_str();
    hipMemcpy(&d_str[i*MTS], c_str, str.size()+1, hipMemcpyHostToDevice);
  }
 
  

  hipMalloc(&tokens, sizeof(int) * TPW * h_vec.size());
  hipMalloc(&values, sizeof(char) * MTS * h_vec.size());
  hipMemset(values, '\0', sizeof(char) * MTS * h_vec.size());
  hipMemset(tokens, -1, sizeof(int) * TPW * h_vec.size());
  htokens = (int *)malloc(sizeof(int) * TPW * h_vec.size());
  hvalues = (char *)malloc(sizeof(char) * MTS * h_vec.size());

  
  int gridX = ((h_vec.size()+BLOCKSIZE-1)/BLOCKSIZE);
  dim3 block(BLOCKSIZE,1,1);
  dim3 grid(gridX,1,1);


  // Start of GPU Processing
  clock_t begin = clock(); 
  init<<<grid,block>>>(d_str, tokens, values, h_vec.size());
  hipMemcpy(htokens, tokens, sizeof(int) * TPW * h_vec.size(), hipMemcpyDeviceToHost);
  hipMemcpy(hvalues, values, sizeof(char) * MTS * h_vec.size(), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  clock_t end = clock();
  // End of GPU Processing  
  generateValues(hvalues,h_vec.size());
  thrust::host_vector<std::string> tokenData = generateValues(hvalues,h_vec.size());
  thrust::host_vector<int> tokensList = generateTokens(htokens,h_vec.size()*TPW);

  double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  printOutput(tokenData,tokensList,time_spent);


  return 0;
}

