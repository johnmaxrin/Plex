#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctype.h>
#include "helpers/enum.h"
#include "helpers/helper.h"
#include "fileops/plexFile.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__global__ void init(char *d_str, int *tokens, char *values, int *size)
{

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  char *data = &d_str[idx*20];
  char *parseData;
  parseData = parse(data, tokens, values,idx);
  int start = idx*20;

  int i=start;
  while(parseData[i]!='\0')
    values[i] = parseData[i++];
  
  printf("%s\n",parseData);

}

int main(int argc, char *argv[])
{

  char *d_str;

  // const char h_str[20] = "{apple";
  thrust::host_vector<std::string> h_vec = readFile(argv[1]);

  hipMalloc(&d_str, h_vec.size() * sizeof(char) * 20);
  for (int i = 0; i < h_vec.size(); ++i)
  {
    std::string str = h_vec[i];
    const char *c_str = str.c_str();
    hipMemcpy(&d_str[i*20], c_str, str.size()+1, hipMemcpyHostToDevice);
  }

  // Total word * 3 tokens we need at max. 2 can be enough though.
  int *htokens, *tokens, *size;
  char *hvalues, *values;

  hipMalloc(&tokens, sizeof(int) * 3 * h_vec.size());
  hipMalloc(&values, sizeof(char) * 20 * h_vec.size());
  hipMemset(values, '\0', sizeof(char) * 20 * h_vec.size());
  hipMemset(tokens, -1, sizeof(int) * 3 * h_vec.size());
  hipMalloc(&size, sizeof(int) * 1);
  htokens = (int *)malloc(sizeof(int) * 3 * h_vec.size());
  hvalues = (char *)malloc(sizeof(char) * 20 * h_vec.size());

  init<<<1, h_vec.size()>>>(d_str, tokens, values, size);

  hipError_t error = hipMemcpy(htokens, tokens, sizeof(int) * 3 * h_vec.size(), hipMemcpyDeviceToHost);
  hipError_t error1 = hipMemcpy(hvalues, values, sizeof(int) * 20 * h_vec.size(), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  for (int i = 0; i < 3*h_vec.size(); ++i)
    printf("%d ", htokens[i]);



  return 0;
}