#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctype.h>
#include <sstream>
#include "helpers/enum.h"
#include "helpers/hostHelp.h"
#include "helpers/global.h"
#include "helpers/helper.h"
#include "fileops/plexFile.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>



__global__ void init(char *d_str, int *tokens, char *values)
{

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  char *data = &d_str[idx*MTS];
  char *parseData;
  parseData = parse(data, tokens, values,idx);
  int start = idx*MTS;

  int i=start,j=0;
  while(parseData[j]!='\0')
    values[i++] = parseData[j++];
  

  }


int main(int argc, char *argv[])
{
  int *htokens, *tokens;
  char *hvalues, *values, *d_str;

  // Reading file  
  thrust::host_vector<std::string> h_vec = readFile(argv[1]);

  hipMalloc(&d_str, h_vec.size() * sizeof(char) * MTS);
  for (int i = 0; i < h_vec.size(); ++i)
  {
    std::string str = h_vec[i];
    const char *c_str = str.c_str();
    hipMemcpy(&d_str[i*MTS], c_str, str.size()+1, hipMemcpyHostToDevice);
  }

  

  hipMalloc(&tokens, sizeof(int) * TPW * h_vec.size());
  hipMalloc(&values, sizeof(char) * MTS * h_vec.size());
  hipMemset(values, '\0', sizeof(char) * MTS * h_vec.size());
  hipMemset(tokens, -1, sizeof(int) * TPW * h_vec.size());
  htokens = (int *)malloc(sizeof(int) * TPW * h_vec.size());
  hvalues = (char *)malloc(sizeof(char) * MTS * h_vec.size());

  // Start of GPU Processing 
  init<<<1, h_vec.size()>>>(d_str, tokens, values);
  hipMemcpy(htokens, tokens, sizeof(int) * TPW * h_vec.size(), hipMemcpyDeviceToHost);
  hipMemcpy(hvalues, values, sizeof(char) * MTS * h_vec.size(), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  // End of GPU Processing 

  generateValues(hvalues,h_vec.size());
  thrust::host_vector<std::string> tokenData = generateValues(hvalues,h_vec.size());
  thrust::host_vector<int> tokensList = generateTokens(htokens,h_vec.size()*TPW);


  for(int i = 0; i<tokensList.size(); ++i)
    printf("< %s , %d >\n",tokenData[i].c_str(),tokensList[i]);
  

  return 0;
}

